
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1000

__global__ void vector_add(int *a, int *b, int *c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int a[N], b[N], c[N];
    int *d_a, *d_b, *d_c;

    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = N - i;
    }

    hipMalloc((void **)&d_a, N * sizeof(int));
    hipMalloc((void **)&d_b, N * sizeof(int));
    hipMalloc((void **)&d_c, N * sizeof(int));

    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    int block_size = 256;
    int grid_size = (N + block_size - 1) / block_size;
    vector_add<<<grid_size, block_size>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("First 10 elements of the result:\n");
    for (int i = 0; i < 10; i++) {
        printf("c[%d] = %d\n", i, c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
