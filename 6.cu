
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void matrix_multiply(int *a, int *b, int *c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n && col < n) {
        int sum = 0;
        for (int k = 0; k < n; k++) {
            sum += a[row * n + k] * b[k * n + col];
        }
        c[row * n + col] = sum;
    }
}

__global__ void matrix_transpose(int *input, int *output, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n && col < n) {
        output[col * n + row] = input[row * n + col];
    }
}

__global__ void matrix_compare(int *a, int *b, int *result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n * n && a[idx] != b[idx]) {
        *result = 0;
    }
}

int main() {
    int a[N][N], b[N][N], c[N][N], ta[N][N], tb[N][N], tc[N][N];
    int *d_a, *d_b, *d_c, *d_ta, *d_tb, *d_tc;
    int size = N * N * sizeof(int);
    int result = 1, *d_result;

    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++) {
            a[i][j] = i + 1;
            b[i][j] = j + 1;
        }

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    hipMalloc(&d_ta, size);
    hipMalloc(&d_tb, size);
    hipMalloc(&d_tc, size);
    hipMalloc(&d_result, sizeof(int));

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_result, &result, sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + 15) / 16, (N + 15) / 16);

    // Transpose A and B
    matrix_transpose<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_ta, N);
    matrix_transpose<<<blocksPerGrid, threadsPerBlock>>>(d_b, d_tb, N);

    // C = A * B
    matrix_multiply<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    // TC = TA * TB
    matrix_multiply<<<blocksPerGrid, threadsPerBlock>>>(d_ta, d_tb, d_tc, N);

    // Compare C and TC
    matrix_compare<<<(N * N + 255) / 256, 256>>>(d_c, d_tc, d_result, N);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    hipMemcpy(tc, d_tc, size, hipMemcpyDeviceToHost);
    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("Matrix C:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }

    printf("Matrix TC:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", tc[i][j]);
        }
        printf("\n");
    }

    printf("C and TC are %sequal.\n", result ? "" : "not ");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_ta);
    hipFree(d_tb);
    hipFree(d_tc);
    hipFree(d_result);

    return 0;
}
