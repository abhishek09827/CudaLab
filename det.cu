#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 3

__global__ void determinant_kernel(float* mat, float* det, int n) {
    int idx = threadIdx.x;
    if (idx >= n) return;

    // Simple row elimination for LU decomposition
    for (int k = 0; k < n; ++k) {
        if (idx > k && mat[k * n + k] != 0) {
            float factor = mat[idx * n + k] / mat[k * n + k];
            for (int j = k; j < n; ++j) {
                mat[idx * n + j] -= factor * mat[k * n + j];
            }
        }
        __syncthreads();
    }

    if (idx == 0) {
        *det = 1.0f;
        for (int i = 0; i < n; ++i) {
            *det *= mat[i * n + i];
        }
    }
}

int main() {
    float h_matrix[N * N] = {4, 1, 2, 1, 5, 3, 2, 3, 6};
    float h_det = 0;
    float *d_matrix, *d_det;

    hipMalloc(&d_matrix, N * N * sizeof(float));
    hipMalloc(&d_det, sizeof(float));

    hipMemcpy(d_matrix, h_matrix, N * N * sizeof(float), hipMemcpyHostToDevice);

    determinant_kernel<<<1, N>>>(d_matrix, d_det, N);
    hipMemcpy(&h_det, d_det, sizeof(float), hipMemcpyDeviceToHost);

    printf("Determinant: %f\n", h_det);

    hipFree(d_matrix);
    hipFree(d_det);
    return 0;
}
