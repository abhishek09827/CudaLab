
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void matrix_add(int *a, int *b, int *c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n && col < n) {
        int idx = row * n + col;
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int a[N][N], b[N][N], c[N][N], *d_a, *d_b, *d_c;
    int size = N * N * sizeof(int);

    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++) {
            a[i][j] = i + j;
            b[i][j] = i - j;
        }
    printf("Matrix A:\n");
    for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
        printf("%d ", a[i][j]);
    }
    printf("\n");
}

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + 15) / 16, (N + 15) / 16);
    matrix_add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printf("Result matrix:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
