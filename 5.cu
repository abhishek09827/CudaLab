
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void matrix_multiply(int *a, int *b, int *c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n && col < n) {
        int sum = 0;
        for (int k = 0; k < n; k++) {
            sum += a[row * n + k] * b[k * n + col];
        }
        c[row * n + col] = sum;
    }
}

int main() {
    int a[N][N], b[N][N], c[N][N], *d_a, *d_b, *d_c;
    int size = N * N * sizeof(int);

    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++) {
            a[i][j] = i + 1;
            b[i][j] = j + 1;
        }

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + 15) / 16, (N + 15) / 16);
    matrix_multiply<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printf("Result matrix:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
