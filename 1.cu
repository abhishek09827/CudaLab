
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main() {
    int a = 5, b = 10, c;
    int *d_a, *d_b, *d_c;

    // Allocate memory on the GPU
    hipMalloc((void **)&d_a, sizeof(int));
    hipMalloc((void **)&d_b, sizeof(int));
    hipMalloc((void **)&d_c, sizeof(int));

    // Copy inputs to GPU
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    add<<<1, 1>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("The sum of %d and %d is %d\n", a, b, c);

    // Free GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
